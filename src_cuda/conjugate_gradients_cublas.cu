#include "hip/hip_runtime.h"
#ifndef CUBLAS_CG_H
#define CUBLAS_CG_H

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdexcept>
#include <utils.cuh>

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

void transfer_to_host(const double * d_x, double * h_x, size_t size)
{
    CUDA_CHECK(hipMemcpy(h_x, d_x, size * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
}

void conjugate_gradients_cublas(const double * h_A, const double * h_b, double * h_x, size_t size, int max_iters, double rel_error)
{
    const double * d_A, * d_b;
    int num_iters;

    double alpha, beta, bb, rr, rr_new;
    double * d_r, * d_p, * d_Ap, * d_x;

    CUDA_CHECK(hipMalloc((void**)&d_A, size * size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_b, size * sizeof(double)));

    CUDA_CHECK(hipMalloc((void**)&d_r, size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_p, size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_Ap, size * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_x, size * sizeof(double)));

    CUDA_CHECK(hipMemcpy((void*)d_A, h_A, size * size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy((void*)d_b, h_b, size * sizeof(double), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemset(d_x, 0, size * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_r, d_b, size * sizeof(double), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_p, d_b, size * sizeof(double), hipMemcpyDeviceToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    const double one = 1.0;
    const double zero = 0.0;

    hipblasDdot(handle, size, d_b, 1, d_b, 1, &bb);
    rr = bb;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        // gemv(1.0, A, p, 0.0, Ap, size, size);
        CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, size, size, &one, d_A, size, d_p, 1, &zero, d_Ap, 1));
        // alpha = rr / dot(p, Ap, size);
        CUBLAS_CHECK(hipblasDdot(handle, size, d_p, 1, d_Ap, 1, &alpha));
        alpha = rr / alpha;
        // axpby(alpha, p, 1.0, x, size);
        CUBLAS_CHECK(hipblasDaxpy(handle, size, &alpha, d_p, 1, d_x, 1));
        // axpby(-alpha, Ap, 1.0, r, size);
        alpha = -alpha;
        CUBLAS_CHECK(hipblasDaxpy(handle, size, &alpha, d_Ap, 1, d_r, 1));
        // rr_new = dot(r, r, size);
        CUBLAS_CHECK(hipblasDdot(handle, size, d_r, 1, d_r, 1, &rr_new));
        beta = rr_new / rr;
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
        // axpby(1.0, r, beta, p, size);
        CUBLAS_CHECK(hipblasDscal(handle, size, &beta, d_p, 1));
        double one = 1.0;
        CUBLAS_CHECK(hipblasDaxpy(handle, size, &one, d_r, 1, d_p, 1));
    }

    CUBLAS_CHECK(hipblasDestroy(handle));

    transfer_to_host(d_x, h_x, size);

    CUDA_CHECK(hipFree((void*)d_A));
    CUDA_CHECK(hipFree((void*)d_b));
    CUDA_CHECK(hipFree(d_r));
    CUDA_CHECK(hipFree(d_p));
    CUDA_CHECK(hipFree(d_Ap));
    CUDA_CHECK(hipFree(d_x));

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}

#endif