#include "hip/hip_runtime.h"
#ifndef MULTI_GPU_LUCA_HPP
#define MULTI_GPU_LUCA_HPP

#include <cstdio>
#include <hip/hip_runtime.h>
#include <utils.cuh>
#include <conjugate_gradients_gpu.cu>
#include <nccl.h>

#define nranks 4
#define TILE_DIM 32
#define BLOCK_ROWS 8

namespace luca {

// ncclComm_t comms[nranks];
// int devs[nranks] = { 0, 1, 2, 3 };


__global__ void transpose(double *odata, const double *idata, int nrows, int ncols)
{
    __shared__ double block[TILE_DIM][TILE_DIM+1];
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    if((x < ncols) && (y < nrows))
    {
        block[threadIdx.y][threadIdx.x] = idata[y*ncols + x];
    }
    __syncthreads();
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    if((x < nrows) && (y < ncols))
    {
        odata[y*nrows + x] = block[threadIdx.x][threadIdx.y];
    }       
}

void gemv_mutli_gpu_tiled_kernel_launcher(const double ** local_A, const double * x, double * y, size_t * num_rows_per_device, size_t num_cols, hipStream_t * s)
{
    int number_of_devices; hipError_t err; /*ncclResult_t nccl_err;*/

    err = hipGetDeviceCount(&number_of_devices); cuda_err_check(err, __FILE__, __LINE__);

    int threadsPerRow = 10;
    size_t sharedMemSize = num_cols / threadsPerRow * sizeof(double);

    double ** y_partial_local = (double**)malloc(number_of_devices * sizeof(double*));
    double ** y_local = (double**)malloc(number_of_devices * sizeof(double*));
    double ** x_local = (double**)malloc(number_of_devices * sizeof(double*));

    for (int i = 0; i < number_of_devices; i++)
    {
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);

        int rowsperblock = 1024;
        // Define the size of the grid and blocks
        dim3 blockDim(1, rowsperblock);
        dim3 gridDim(threadsPerRow, (num_rows_per_device[i] + rowsperblock - 1) / rowsperblock);

        err = hipMallocAsync((void**)&y_partial_local[i], num_rows_per_device[i] * threadsPerRow * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&y_local[i], num_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&x_local[i], num_cols * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemsetAsync(y_partial_local[i], 0, num_rows_per_device[i] * threadsPerRow * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpyAsync(x_local[i], x, num_cols * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);

        // Launch the kernel
        gemv_tiled_kernel<<<gridDim, blockDim, sharedMemSize, s[i]>>>(local_A[i], x_local[i], y_partial_local[i], num_rows_per_device[i], num_cols);
        reduce_rows<<<(num_rows_per_device[i] + threadsPerRow - 1) / threadsPerRow, threadsPerRow, 0, s[i]>>>(y_partial_local[i], y_local[i], num_rows_per_device[i], threadsPerRow);
    
        if (i > 0)
            {err = hipMemcpyPeerAsync(y + i * (num_rows_per_device[i]), 0, y_local[i], i, num_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);}
        else
            {err = hipMemcpyAsync(y, y_local[i], num_rows_per_device[i] * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);}
    }

    for (int i = 0; i < number_of_devices; i++)
    {
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFreeAsync(y_partial_local[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFreeAsync(y_local[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);
    }

    // for (int i = 0; i < number_of_devices; i++)
    // {
    //     err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
    // }

    // sync all streams
    // for(int i = 0; i < number_of_devices; i++) err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__);

    err = hipSetDevice(0); cuda_err_check(err, __FILE__, __LINE__);

    free(y_partial_local);
    free(y_local);
    free(x_local);
}



void par_conjugate_gradients_multi_gpu(const double * h_A, const double * h_b, double * h_x, size_t size, int max_iters, double rel_error)
{
    hipError_t err;

    // nccl_err = ncclCommInitAll(comms, nranks, devs); nccl_err_check(nccl_err, __FILE__, __LINE__);

    const double /* d_A,*/ * d_b;
    int num_iters;

    double alpha, beta, bb, rr, rr_new;
    double * d_r, * d_p, * d_Ap, * d_x;

    int number_of_devices;
    hipStream_t * s;
    size_t * number_of_rows_per_device;
    const double ** d_local_A, ** d_local_A_transposed;

    err = hipGetDeviceCount(&number_of_devices); cuda_err_check(err, __FILE__, __LINE__);
    s = (hipStream_t*)malloc(number_of_devices * sizeof(hipStream_t));
    d_local_A = (const double**)malloc(number_of_devices * sizeof(const double*));
    d_local_A_transposed = (const double**)malloc(number_of_devices * sizeof(double*));
    number_of_rows_per_device = (size_t*)malloc(number_of_devices * sizeof(size_t));

    for(int i = 0; i < number_of_devices; i++)
    {   
        number_of_rows_per_device[i] = (i == number_of_devices - 1) ? size - i * (size / number_of_devices) : size / number_of_devices;
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&d_local_A[i], size * number_of_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&d_local_A_transposed[i], size * number_of_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpyAsync((void*)(d_local_A[i]), h_A + i * (size / number_of_devices) * size, size * number_of_rows_per_device[i] * sizeof(double), hipMemcpyHostToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
        // err = hipMemcpyAsync((void*)d_local_A_transposed[i], (void*)d_local_A[i], size * number_of_rows_per_device[i] * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
        transpose<<<dim3(size / TILE_DIM + 1, size / TILE_DIM + 1), dim3(TILE_DIM, TILE_DIM), 0, s[i]>>>((double*)d_local_A_transposed[i], d_local_A[i], number_of_rows_per_device[i], size);
    }
    
    err = hipSetDevice(0); cuda_err_check(err, __FILE__, __LINE__);

    // err = hipMalloc((void**)&d_A, size * size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_b, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMalloc((void**)&d_r, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_p, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_Ap, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_x, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);

    // err = hipMemcpy((void*)d_A, h_A, size * size * sizeof(double), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy((void*)d_b, h_b, size * sizeof(double), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemset(d_x, 0, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_r, d_b, size * sizeof(double), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_p, d_b, size * sizeof(double), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);

    // sync all streams
    for(int i = 0; i < number_of_devices; i++) { err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__); err = hipFreeAsync((void*)d_local_A[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);}

    bb = dot_kernel_launcher(d_b, d_b, size);
    rr = bb;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);
        // gemv(1.0, A, p, 0.0, Ap, size, size);
        gemv_mutli_gpu_tiled_kernel_launcher(d_local_A_transposed, d_p, d_Ap, number_of_rows_per_device, size, s);
        // gemv_kernel_launcher(1.0, d_A, d_p, 0.0, d_Ap, size, size);
        // alpha = rr / dot(p, Ap, size);
        alpha = rr / dot_kernel_launcher(d_p, d_Ap, size);
        // axpby(alpha, p, 1.0, x, size);
        axpby_kernel_launcher(alpha, d_p, 1.0, d_x, size);
        // axpby(-alpha, Ap, 1.0, r, size);
        axpby_kernel_launcher(-alpha, d_Ap, 1.0, d_r, size);
        // rr_new = dot(r, r, size);
        rr_new = dot_kernel_launcher(d_r, d_r, size);
        beta = rr_new / rr;
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
        // axpby(1.0, r, beta, p, size);
        axpby_kernel_launcher(1.0, d_r, beta, d_p, size);
    }

    transfer_to_host(d_x, h_x, size);

    // err = hipFree((void*)d_A); cuda_err_check(err, __FILE__, __LINE__);
    for (int i = 0; i < number_of_devices; i++)
    {
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree((void*)d_local_A_transposed[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamDestroy(s[i]); cuda_err_check(err, __FILE__, __LINE__);
    }
    err = hipFree((void*)d_b); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_r); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_p); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_Ap); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_x); cuda_err_check(err, __FILE__, __LINE__);

    free(s);
    free(d_local_A);
    free(d_local_A_transposed);
    free(number_of_rows_per_device);

    // for (int i=0; i<nranks; i++)
    //     ncclCommDestroy(comms[i]);

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}
}

#endif