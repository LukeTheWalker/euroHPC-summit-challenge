#include "hip/hip_runtime.h"
#ifndef MULTI_GPU_NCCL_LUCA_HPP
#define MULTI_GPU_NCCL_LUCA_HPP

#include <cstdio>
#include <hip/hip_runtime.h>
#include <utils.cuh>
#include <conjugate_gradients_gpu.cu>
#include <conjugate_gradients_mutli_gpu.cu>
#include <nccl.h>
#include <omp.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define NDEVICES_PER_NODE 4


namespace luca {

ncclUniqueId id;
ncclComm_t comms[NDEVICES_PER_NODE];
int myRank, nRanks, localRank = 0;

void initialize_nccl () {

    int mpi_err; ncclResult_t nccl_err; hipError_t cuda_err;

    // get localRank
    mpi_err = MPI_Comm_rank(MPI_COMM_WORLD, &myRank); mpi_err_check(mpi_err, __FILE__, __LINE__);
    mpi_err = MPI_Comm_size(MPI_COMM_WORLD, &nRanks); mpi_err_check(mpi_err, __FILE__, __LINE__);

    // calculating localRank which is used in selecting a GPU
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    mpi_err = MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD); mpi_err_check(mpi_err, __FILE__, __LINE__);
    for (int p = 0; p < nRanks; p++)
    {
        if (p == myRank)
            break;
        if (hostHashs[p] == hostHashs[myRank])
            localRank++;
    }

    if (myRank == 0)
        ncclGetUniqueId(&id);
    mpi_err = MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD); mpi_err_check(mpi_err, __FILE__, __LINE__);

    nccl_err = ncclGroupStart(); nccl_err_check(nccl_err, __FILE__, __LINE__);
    for (int i = 0; i < NDEVICES_PER_NODE; i++)
    {
        cuda_err = hipSetDevice(localRank * NDEVICES_PER_NODE + i); cuda_err_check(cuda_err, __FILE__, __LINE__);
        nccl_err = ncclCommInitRank(comms + i, nRanks * NDEVICES_PER_NODE, id, myRank * NDEVICES_PER_NODE + i); nccl_err_check(nccl_err, __FILE__, __LINE__);
    }
    nccl_err = ncclGroupEnd(); nccl_err_check(nccl_err, __FILE__, __LINE__);
    fprintf(stderr,"[MPI Rank %d] responsible for GPU %d-%d\n", myRank, myRank * NDEVICES_PER_NODE, myRank * NDEVICES_PER_NODE + NDEVICES_PER_NODE - 1);
}

void gemv_mutli_gpu_nccl_tiled_kernel_launcher(const double ** local_A, const double * x, double * y, size_t * num_rows_per_device, size_t * num_rows_per_node, size_t num_cols, hipStream_t * s)
{
    int number_of_devices; hipError_t err; /*ncclResult_t nccl_err;*/

    err = hipGetDeviceCount(&number_of_devices); cuda_err_check(err, __FILE__, __LINE__);

    int threadsPerRow = 10;
    size_t sharedMemSize = num_cols / threadsPerRow * sizeof(double);

    double ** y_partial_local = (double**)malloc(number_of_devices * sizeof(double*));
    double ** y_local = (double**)malloc(number_of_devices * sizeof(double*));
    double ** x_local = (double**)malloc(number_of_devices * sizeof(double*));

    for (int i = 0; i < number_of_devices; i++)
    {
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);

        int rowsperblock = 1024;
        // Define the size of the grid and blocks
        dim3 blockDim(1, rowsperblock);
        dim3 gridDim(threadsPerRow, (num_rows_per_device[i] + rowsperblock - 1) / rowsperblock);

        err = hipMallocAsync((void**)&y_partial_local[i], num_rows_per_device[i] * threadsPerRow * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&y_local[i], num_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&x_local[i], num_cols * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemsetAsync(y_partial_local[i], 0, num_rows_per_device[i] * threadsPerRow * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpyAsync(x_local[i], x, num_cols * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);

        // Launch the kernel
        gemv_tiled_kernel<<<gridDim, blockDim, sharedMemSize, s[i]>>>(local_A[i], x_local[i], y_partial_local[i], num_rows_per_device[i], num_cols);
        reduce_rows<<<(num_rows_per_device[i] + threadsPerRow - 1) / threadsPerRow, threadsPerRow, 0, s[i]>>>(y_partial_local[i], y_local[i], num_rows_per_device[i], threadsPerRow);
    
        // err = hipMemcpyAsync(y + i * (num_rows_per_device[i]), y_local[i], num_rows_per_device[i] * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
    }

    // for (int i = 0; i < number_of_devices; i++)
    // {
    //     err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
    //     err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__);
    //     err = hipFreeAsync(y_partial_local[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);
    //     err = hipFreeAsync(y_local[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);
    // }

    ncclResult_t nccl_err;

    nccl_err = ncclGroupStart(); nccl_err_check(nccl_err, __FILE__, __LINE__);
    // device 0 on rank 0 receives all the pieces of the array
    for (int i = 0; i < number_of_devices; i++)
        nccl_err = ncclSend(y_local[i], num_rows_per_device[i], ncclDouble, 0, comms[i], s[i]); nccl_err_check(nccl_err, __FILE__, __LINE__);
    
    if (myRank == 0){
        int progressive_offset = 0;
        for (int r = 0; r < nRanks; r++){
            for (int i = 0; i < number_of_devices; i++){
                int num_to_transfer = (i == number_of_devices - 1) ? num_rows_per_node[r] - i * (num_rows_per_node[r] / number_of_devices) : num_rows_per_node[r] / number_of_devices;
                nccl_err = ncclRecv(y + progressive_offset, num_to_transfer, ncclDouble, r * NDEVICES_PER_NODE + i, comms[i], s[i]); nccl_err_check(nccl_err, __FILE__, __LINE__);
                progressive_offset += num_to_transfer;
            }
        }
    }
    nccl_err = ncclGroupEnd(); nccl_err_check(nccl_err, __FILE__, __LINE__);

    // for (int i = 0; i < number_of_devices; i++)
    // {
    //     err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
    // }

    // sync all streams
    // for(int i = 0; i < number_of_devices; i++) err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__);

    err = hipSetDevice(0); cuda_err_check(err, __FILE__, __LINE__);

    free(y_partial_local);
    free(y_local);
    free(x_local);
}



void par_conjugate_gradients_multi_gpu_nccl(const double * h_A, const double * h_b, double * h_x, size_t size, int max_iters, double rel_error)
{
    initialize_nccl();
    hipError_t err;

    // nccl_err = ncclCommInitAll(comms, nranks, devs); nccl_err_check(nccl_err, __FILE__, __LINE__);

    int number_of_devices;
    hipStream_t * s;
    size_t * number_of_rows_per_device, *number_of_rows_per_node;
    const double ** d_local_A, ** d_local_A_transposed;

    err = hipGetDeviceCount(&number_of_devices); cuda_err_check(err, __FILE__, __LINE__);
    s = (hipStream_t*)malloc(number_of_devices * sizeof(hipStream_t));
    d_local_A = (const double**)malloc(number_of_devices * sizeof(const double*));
    d_local_A_transposed = (const double**)malloc(number_of_devices * sizeof(double*));
    number_of_rows_per_device = (size_t*)malloc(number_of_devices * sizeof(size_t));
    number_of_rows_per_node = (size_t*)malloc(sizeof(size_t) * nRanks);


    // int number_of_rows_per_node = (myRank == nRanks - 1) ? size - myRank * (size / nRanks) : size / nRanks;
    for (int i = 0; i < nRanks; i++) number_of_rows_per_node[i] = (i == nRanks - 1) ? size - i * (size / nRanks) : size / nRanks;


    omp_set_num_threads(number_of_devices);

    #pragma omp parallel for
    for(int i = 0; i < number_of_devices; i++)
    {
        // number_of_rows_per_device[i] = (i == number_of_devices - 1) ? size - i * (size / number_of_devices) : size / number_of_devices;
        number_of_rows_per_device[i] = (i == number_of_devices - 1) ? number_of_rows_per_node[i] - i * (number_of_rows_per_node[i] / number_of_devices) : number_of_rows_per_node[i] / number_of_devices;
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&d_local_A[i], size * number_of_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMallocAsync((void**)&d_local_A_transposed[i], size * number_of_rows_per_device[i] * sizeof(double), s[i]); cuda_err_check(err, __FILE__, __LINE__);
        // err = hipMemcpyAsync((void*)(d_local_A[i]), h_A + i * (size / number_of_devices) * size, size * number_of_rows_per_device[i] * sizeof(double), hipMemcpyHostToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
        // each node does size / nRanks and each device does size / nRanks / nDevices
        err = hipMemcpyAsync((void*)(d_local_A[i]), h_A + myRank * (size / nRanks) + i * (size / nRanks / number_of_devices) * size, size * number_of_rows_per_device[i] * sizeof(double), hipMemcpyHostToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
        // err = hipMemcpyAsync((void*)d_local_A_transposed[i], (void*)d_local_A[i], size * number_of_rows_per_device[i] * sizeof(double), hipMemcpyDeviceToDevice, s[i]); cuda_err_check(err, __FILE__, __LINE__);
        transpose<<<dim3(size / TILE_DIM + 1, size / TILE_DIM + 1), dim3(TILE_DIM, TILE_DIM), 0, s[i]>>>((double*)d_local_A_transposed[i], d_local_A[i], number_of_rows_per_device[i], size);
    }

    const double /* d_A,*/ * d_b;
    int num_iters;

    double alpha = 0, beta = 0, rr = 0, rr_new = 0, bb = 0;
    double * d_r, * d_p, * d_Ap, * d_x;

    if (myRank == 0)
    {
        err = hipSetDevice(0); cuda_err_check(err, __FILE__, __LINE__);

        // err = hipMalloc((void**)&d_A, size * size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_b, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMalloc((void**)&d_r, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_p, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_Ap, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_x, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);

        // err = hipMemcpy((void*)d_A, h_A, size * size * sizeof(double), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy((void*)d_b, h_b, size * sizeof(double), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemset(d_x, 0, size * sizeof(double)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_r, d_b, size * sizeof(double), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_p, d_b, size * sizeof(double), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);
    }

    // sync all streams
    for(int i = 0; i < number_of_devices; i++) { err = hipStreamSynchronize(s[i]); cuda_err_check(err, __FILE__, __LINE__); err = hipFreeAsync((void*)d_local_A[i], s[i]); cuda_err_check(err, __FILE__, __LINE__);}

    if (myRank == 0)
    {
        bb = dot_kernel_launcher(d_b, d_b, size);
        rr = bb;
    }
    bool done = false; int mpi_err;
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        mpi_err = MPI_Bcast(&done, 1, MPI_C_BOOL, 0, MPI_COMM_WORLD); mpi_err_check(mpi_err, __FILE__, __LINE__);
        if (done) { break; }
        // err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);
        // gemv(1.0, A, p, 0.0, Ap, size, size);
        gemv_mutli_gpu_nccl_tiled_kernel_launcher(d_local_A_transposed, d_p, d_Ap, number_of_rows_per_device, number_of_rows_per_node, size, s);
        // gemv_kernel_launcher(1.0, d_A, d_p, 0.0, d_Ap, size, size);
        // alpha = rr / dot(p, Ap, size);
        if (myRank == 0) {
            alpha = rr / dot_kernel_launcher(d_p, d_Ap, size);
            // axpby(alpha, p, 1.0, x, size);
            axpby_kernel_launcher(alpha, d_p, 1.0, d_x, size);
            // axpby(-alpha, Ap, 1.0, r, size);
            axpby_kernel_launcher(-alpha, d_Ap, 1.0, d_r, size);
            // rr_new = dot(r, r, size);
            rr_new = dot_kernel_launcher(d_r, d_r, size);
            beta = rr_new / rr;
            rr = rr_new;
            if(std::sqrt(rr / bb) < rel_error) { done = true; }
            // axpby(1.0, r, beta, p, size);
            axpby_kernel_launcher(1.0, d_r, beta, d_p, size);
        }
    }

    if (myRank == 0)
        transfer_to_host(d_x, h_x, size);

    for (int i = 0; i < number_of_devices; i++)
        nccl_err_check(ncclCommDestroy(comms[i]), __FILE__, __LINE__);

    // err = hipFree((void*)d_A); cuda_err_check(err, __FILE__, __LINE__);
    for (int i = 0; i < number_of_devices; i++)
    {
        err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree((void*)d_local_A_transposed[i]); cuda_err_check(err, __FILE__, __LINE__);
        err = hipStreamDestroy(s[i]); cuda_err_check(err, __FILE__, __LINE__);
    }

    if (myRank == 0){    
        err = hipFree((void*)d_b); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_r); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_p); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_Ap); cuda_err_check(err, __FILE__, __LINE__);
        err = hipFree(d_x); cuda_err_check(err, __FILE__, __LINE__);
    }
    free(s);
    free(d_local_A);
    free(d_local_A_transposed);
    free(number_of_rows_per_device);

    // for (int i=0; i<nranks; i++)
    //     ncclCommDestroy(comms[i]);

    if (myRank == 0){
        if(num_iters <= max_iters)
        {
            printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
        }
        else
        {
            printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
        }
    }
}
}

#endif