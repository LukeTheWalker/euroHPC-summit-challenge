#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <chrono>
#include <conjugate_gradients_gpu.cu>
#include <conjugate_gradients_cpu_serial.hpp>
#include <conjugate_gradients_cpu_openmp.hpp>
#include <conjugate_gradients_gpu_tommy.cu>
#include <conjugate_gradients_multi_gpu.cu>
#include <conjugate_gradients_multi_gpu_nccl.cu>
#include <conjugate_gradients_cublas.cu>
#include <utils.cuh>
#include <functional>
#include <string>

int main(int argc, char ** argv)
{
    MPI_Init(&argc, &argv);
    // printf("Usage: ./random_matrix input_file_matrix.bin input_file_rhs.bin output_file_sol.bin max_iters rel_error\n");
    // printf("All parameters are optional and have default values\n");
    // printf("\n");

    const char * input_file_matrix = "io/matrix.bin";
    const char * input_file_rhs = "io/rhs.bin";
    const char * output_file_sol = "io/sol.bin";
    int max_iters = 1000;
    double rel_error = 1e-9;

    if(argc > 1) input_file_matrix = argv[1];
    if(argc > 2) input_file_rhs = argv[2];
    if(argc > 3) output_file_sol = argv[3];
    if(argc > 4) max_iters = atoi(argv[4]);
    if(argc > 5) rel_error = atof(argv[5]);

    // printf("Command line arguments:\n");
    // printf("  input_file_matrix: %s\n", input_file_matrix);
    // printf("  input_file_rhs:    %s\n", input_file_rhs);
    // printf("  output_file_sol:   %s\n", output_file_sol);
    // printf("  max_iters:         %d\n", max_iters);
    // printf("  rel_error:         %e\n", rel_error);
    // printf("\n");

    double * matrix;
    double * rhs;
    size_t size;

    {
        // printf("Reading matrix from file ...\n");
        size_t matrix_rows;
        size_t matrix_cols;
        bool success_read_matrix = read_matrix_from_file(input_file_matrix, &matrix, &matrix_rows, &matrix_cols);
        if(!success_read_matrix)
        {
            fprintf(stderr, "Failed to read matrix\n");
            return 1;
        }
        // printf("Done\n");
        // printf("\n");

        // printf("Reading right hand side from file ...\n");
        size_t rhs_rows;
        size_t rhs_cols;
        bool success_read_rhs = read_matrix_from_file(input_file_rhs, &rhs, &rhs_rows, &rhs_cols);
        if(!success_read_rhs)
        {
            fprintf(stderr, "Failed to read right hand side\n");
            return 2;
        }
        // printf("Done\n");
        // printf("\n");

        if(matrix_rows != matrix_cols)
        {
            fprintf(stderr, "Matrix has to be square\n");
            return 3;
        }
        if(rhs_rows != matrix_rows)
        {
            fprintf(stderr, "Size of right hand side does not match the matrix\n");
            return 4;
        }
        if(rhs_cols != 1)
        {
            fprintf(stderr, "Right hand side has to have just a single column\n");
            return 5;
        }

        size = matrix_rows;
    }

    int number_of_tests = 1;
    int times[number_of_tests];
    double sol [number_of_tests][size];
    std::function<void(double*, double*, double*, size_t, int, double)> implementations_to_test[number_of_tests] = 
    {luca::par_conjugate_gradients_multi_gpu_nccl};
    std::string names[number_of_tests] = {"NCCL"};

    int impl_used = argc > 6 ? atoi(argv[6]) : 0;

    {
        int number_of_gpus; hipError_t err;
        err = hipGetDeviceCount(&number_of_gpus); cuda_err_check(err, __FILE__, __LINE__);
        for (int i = 0; i < number_of_gpus; i++)
        {
            err = hipSetDevice(i); cuda_err_check(err, __FILE__, __LINE__);
            err = hipFree(0); cuda_err_check(err, __FILE__, __LINE__);
        }

        initialize_nccl();

        int rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);

        long start_time = 0, end_time = 0;

        if (rank == 0){
            times[impl_used] = 0;
            printf("Solving the system with %s ...\n", names[impl_used].c_str());
            start_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        }
        MPI_Barrier(MPI_COMM_WORLD);
        implementations_to_test[impl_used](matrix, rhs, sol[impl_used], size, max_iters, rel_error);
        if (rank == 0){
            end_time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
            times[impl_used] = (end_time - start_time);
            printf("Done in %f milliseconds\n", times[impl_used] / 1000.0);
        }

        MPI_Finalize();
        if (rank != 0) return 0;
    }

    
    // printf("Writing solution to file ...\n");
    bool success_write_sol = write_matrix_to_file(output_file_sol, sol[impl_used], size, 1);
    if(!success_write_sol)
    {
        fprintf(stderr, "Failed to save solution\n");
        return 6;
    }

    FILE * time_f = fopen("output/time.txt", "w");
    fprintf(time_f, "%d", time);
    fclose(time_f);
   
    hipError_t err;
    err = hipHostFree(matrix); cuda_err_check(err, __FILE__, __LINE__);
    err = hipHostFree(rhs); cuda_err_check(err, __FILE__, __LINE__);

    printf("Finished successfully\n\n");

    return 0;
}
